#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int BlockSize = 2;
const int ThreadNum = 5;
const int N = 10;

__global__ void increment_gpu(int a[],const int b, const int N)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<N)
         a[idx]=a[idx]+b;
}

int main()
{
    int a[N];
	int *d_a = 0;
	int i = 0;
	const int b = 10;
	
	cout << "a[N] array before scaling: [";
	for(i=0;i<N;i++)
	{
		a[i] = i;
		cout << a[i] << " ";
	}
	cout << "]" << endl;
	
	hipMalloc((void**) &d_a,sizeof(int)*N);
	hipMemcpy(d_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    increment_gpu<<<dimGrid,dimBlock>>>(d_a,b,N);
	hipDeviceSynchronize();
	
	hipMemcpy(a,d_a,sizeof(int)*N,hipMemcpyDeviceToHost);
	
	cout << "a[N] array after scaling: [";
	for(i=0;i<N;i++)
	{
		cout << a[i] << " ";
	}
	cout << "]" << endl;
	
	return 0;
}
