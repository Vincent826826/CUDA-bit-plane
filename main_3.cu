#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> 
#include <stdio.h>
#include <time.h>
#include "parameter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 2;
const int ThreadNum = 8;

// change the way of getting target bit
__device__ int get_target_bit(int byteFlag, int whichBit)
{
    if (whichBit >= 0 && whichBit < 8)
        return (byteFlag & (1<<whichBit)) >> whichBit;
    else
        return 0;
}

__global__ void gpu_bit_plane(int *d_original, int*d_result)
{

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int start_idx = int(idx / 8);
	int bit = idx % 8;
	// Idx 0~7 start from 0
	// Idx 8~15 start from 1
	for(int i = start_idx; i < ARRAY_SIZE; i+=2)
	{
		d_result[i * BYTE_SIZE + bit] = get_target_bit(d_original[i], bit);
	}
}

void generate_number(int *original)
{
    
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        original[i] = int(rand()% (1<<BYTE_SIZE) );
    }
}

void print_original(int* original)
{
	cout<<"Original = "<<endl;
    for(int i = 0; i < ARRAY_SIZE; i++)
        cout<<original[i]<<" ";
    cout<<endl;
}

void print_result1D(int* result)
{
	cout<<"Result = "<<endl;
	for(int i = 0; i < ARRAY_SIZE; i++)
	{
		cout<<"["<<i<<"] : ";
		int sum = 0;
		for(int bit = BYTE_SIZE - 1; bit >= 0; bit--)
		{
			cout<<result[ i * BYTE_SIZE + bit];
			sum += result[i * BYTE_SIZE + bit] << bit;
		}
		cout<<" = "<<sum<<endl;
	}
	cout<<endl;
}

bool validate(int *original, int *result)
{
    // check the result before and after bit plane
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        int sum = 0;
        for(int bit = 0; bit < BYTE_SIZE; bit++)
        {
            sum += result[i * BYTE_SIZE + bit] << bit;
        }
        if(original[i] != sum)
		{
			cout<<"["<<i<<"]"<<" is incorrect!"<<endl;
			return false;
		}
    }
    return true;
}

int main()
{
    int *original =  (int*)calloc(ARRAY_SIZE, sizeof(int));
	int *d_original = 0;
	int *result  = (int*)calloc(BYTE_SIZE*ARRAY_SIZE, sizeof(int));
	int *d_result = 0;

	generate_number(original);

	// init data
    cout<<endl<<"Init finished"<<endl;
    cout<<"Array size is "<<ARRAY_SIZE<<endl;

	clock_t tStart = clock();
	
	hipMalloc((void**) &d_original, sizeof(int)*ARRAY_SIZE);
	hipMemcpy(d_original, original, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &d_result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE);
	hipMemcpy(d_result, result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE, hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    gpu_bit_plane<<<dimGrid,dimBlock>>>(d_original, d_result);
	hipDeviceSynchronize();
	
	hipMemcpy(result, d_result,sizeof(int)*ARRAY_SIZE*BYTE_SIZE,hipMemcpyDeviceToHost);

	printf("Time taken: %.8fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	print_original(original);
	print_result1D(result);

	cout<<"Compare result is ";
    if(validate(original, result))
        cout<<"correct"<<endl;
    else
        cout<<"incorrect"<<endl;

	return 0;
}
