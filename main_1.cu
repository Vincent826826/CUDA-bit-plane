#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> 
#include <stdio.h>
#include <time.h>
#include "parameter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 2;
const int ThreadNum = 8;

__global__ void gpu_bit_plane(int *d_original, int*d_result)
{

	int TotalThread = gridDim.x*blockDim.x;
	int stripe = ARRAY_SIZE / TotalThread;
    int head = (blockIdx.x*blockDim.x + threadIdx.x)*stripe;
	for(int idx = head; idx<(head+stripe); idx++)
	{
		int val_cpy = d_original[idx];
		for(int bit = 0; bit < BYTE_SIZE; bit++)
		{
			d_result[BYTE_SIZE*idx + bit] = val_cpy & 1;
			val_cpy = val_cpy >> 1;
		}
	}
}

void generate_number(int *original)
{
    
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        original[i] = int(rand()% (1<<BYTE_SIZE) );
    }
}

void print_original(int* original)
{
	cout<<"Original = "<<endl;
    for(int i = 0; i < ARRAY_SIZE; i++)
        cout<<original[i]<<" ";
    cout<<endl;
}

void print_result1D(int* result)
{
	cout<<"Result = "<<endl;
	for(int i = 0; i < ARRAY_SIZE; i++)
	{
		cout<<"["<<i<<"] : ";
		int sum = 0;
		for(int bit = BYTE_SIZE - 1; bit >= 0; bit--)
		{
			cout<<result[ i * BYTE_SIZE + bit];
			sum += result[i * BYTE_SIZE + bit] << bit;
		}
		cout<<" = "<<sum<<endl;
	}
	cout<<endl;
}

bool validate(int *original, int *result)
{
    // check the result before and after bit plane
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        int sum = 0;
        for(int bit = 0; bit < BYTE_SIZE; bit++)
        {
            sum += result[i * BYTE_SIZE + bit] << bit;
        }
        if(original[i] != sum)
		{
			cout<<"["<<i<<"]"<<" is incorrect!"<<endl;
			return false;
		}
    }
    return true;
}

int main()
{
    int *original =  (int*)calloc(ARRAY_SIZE, sizeof(int));
	int *d_original = 0;
	int *result  = (int*)calloc(BYTE_SIZE*ARRAY_SIZE, sizeof(int));
	int *d_result = 0;

	generate_number(original);

	// init data
    cout<<endl<<"Init finished"<<endl;
    cout<<"Array size is "<<ARRAY_SIZE<<endl;

	clock_t tStart = clock();
	
	hipMalloc((void**) &d_original, sizeof(int)*ARRAY_SIZE);
	hipMemcpy(d_original, original, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &d_result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE);
	hipMemcpy(d_result, result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE, hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    gpu_bit_plane<<<dimGrid,dimBlock>>>(d_original, d_result);
	hipDeviceSynchronize();
	
	hipMemcpy(result, d_result,sizeof(int)*ARRAY_SIZE*BYTE_SIZE,hipMemcpyDeviceToHost);

	printf("Time taken: %.8fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	print_original(original);
	print_result1D(result);

	cout<<"Compare result is ";
    if(validate(original, result))
        cout<<"correct"<<endl;
    else
        cout<<"incorrect"<<endl;

	return 0;
}
