#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> 
#include <stdio.h>
#include <time.h>
#include "parameter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 1;
const int ThreadNum = 3;

__global__ void gpu_bit_plane(int *d_oringinal, int*d_result)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < BYTE_SIZE; i++)
	{
		d_result[BYTE_SIZE*idx + i] = d_oringinal[idx] + i;
	}

}

void generate_number(int *original)
{
    
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        original[i] = int(rand()% (1<<BYTE_SIZE) );
    }
}

int main()
{
    int *original =  (int*)calloc(ARRAY_SIZE, sizeof(int));
	int *d_oringinal = 0;
	int *result  = (int*)calloc(BYTE_SIZE*ARRAY_SIZE, sizeof(int));
	int *d_result = 0;

	generate_number(original);

	// init data
    cout<<endl<<"Init finished"<<endl;
    cout<<"Array size is "<<ARRAY_SIZE<<endl;

	clock_t tStart = clock();
	
	hipMalloc((void**) &d_oringinal, sizeof(int)*ARRAY_SIZE);
	hipMemcpy(d_oringinal, original, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &d_result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE);
	hipMemcpy(d_result, result, sizeof(int)*ARRAY_SIZE*BYTE_SIZE, hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    gpu_bit_plane<<<dimGrid,dimBlock>>>(d_oringinal, d_result);
	hipDeviceSynchronize();
	
	hipMemcpy(result, d_result,sizeof(int)*ARRAY_SIZE*BYTE_SIZE,hipMemcpyDeviceToHost);
	
	printf("Time taken: %.8fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
	
	return 0;
}
