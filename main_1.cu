#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> 
#include <stdio.h>
#include <time.h>
#include "parameter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 2;
const int ThreadNum = 5;
const int N = 10;

__global__ void increment_gpu(int a[],const int b, const int N)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<N)
         a[idx]=a[idx]+b;
}
__global__ void gpu_bit_plane(int *original, int **result)
{
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        int val_cpy = original[i];
        for(int bit = 0; bit < BYTE_SIZE; bit++)
        {
            result[i][bit] = val_cpy & 1;
            val_cpy = val_cpy >> 1;
        }
    }
}
void generate_number(int *original)
{
    
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        original[i] = int(rand()% (1<<BYTE_SIZE) );
    }
}

bool validate(int *original, int **result)
{
    // check the result before and after bit plane
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        int sum = 0;
        for(int bit = 0; bit < BYTE_SIZE; bit++)
        {
            sum += result[i][bit] << bit;
        }
        if(original[i] != sum)return false;
    }
    return true;
}
void print_original(int* original)
{
    for(int i = 0; i < ARRAY_SIZE; i++)
        cout<<original[i]<<" ";
    cout<<endl;
}
void print_result(int** result)
{
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        for(int j = BYTE_SIZE - 1; j >= 0 ; j--)
        {
            // little-endian represntation
            cout<<result[i][j];
        }
        cout<<endl;
    }
    cout<<endl;
}
int main()
{
    int *original =  (int*)calloc(ARRAY_SIZE, sizeof(int));
    int **result = (int**)calloc(ARRAY_SIZE, sizeof(int*));
    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        result[i] = (int*)calloc(BYTE_SIZE, sizeof(int));
    }

    generate_number(original);
	
	int a[N];
	int *d_a = 0;
	int i = 0;
	const int b = 10;
	
	cout << "a[N] array before scaling: [";
	for(i=0;i<N;i++)
	{
		a[i] = i;
		cout << a[i] << " ";
	}
	cout << "]" << endl;
	
	hipMalloc((void**) &d_a,sizeof(int)*N);
	hipMemcpy(d_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    increment_gpu<<<dimGrid,dimBlock>>>(d_a,b,N);
	hipDeviceSynchronize();
	
	hipMemcpy(a,d_a,sizeof(int)*N,hipMemcpyDeviceToHost);
	
	cout << "a[N] array after scaling: [";
	for(i=0;i<N;i++)
	{
		cout << a[i] << " ";
	}
	cout << "]" << endl;
	
	return 0;
}
