#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> 
#include <stdio.h>
#include <time.h>
#include "parameter.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 2;
const int ThreadNum = 4;

__global__ void test(int *original, int **result)
{

    int TotalThread = blockDim.x * gridDim.x;
    int stripe = ARRAY_SIZE / TotalThread;
    int head = (blockIdx.x * blockDim.x + threadIdx.x) * stripe;
    int LoopLim = head + stripe;

    for(int i = head; i < LoopLim; i++)
    {
        int val_cpy = original[i];
        for(int bit = 0; bit < BYTE_SIZE; bit++)
        {
            result[i][bit] = val_cpy & 1;
            val_cpy = val_cpy >> 1;
        }
    }
}


int main()
{
    int **result;
    int **d_result;
   
	hipMalloc((void**) &d_a,sizeof(int)*N);
	hipMemcpy(d_a,a,sizeof(int)*N,hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    increment_gpu<<<dimGrid,dimBlock>>>(d_a,b,N);
	hipDeviceSynchronize();
	
	hipMemcpy(a,d_a,sizeof(int)*N,hipMemcpyDeviceToHost);
	
	return 0;
}
