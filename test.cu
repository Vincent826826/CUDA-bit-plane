#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int BlockSize = 1;
const int ThreadNum = 3;
const int ARRAY_SIZE = 3;

__global__ void test(int *d_oringinal, int*d_result)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    for(int i = 0; i < 8; i++)
	{
		d_result[8*idx + i] = d_oringinal[idx] + i;
	}

}

int main()
{
    int oringinal[ARRAY_SIZE] = {10, 20, 30};
	int *d_oringinal = 0;
	int *result  = (int*)calloc(8*ARRAY_SIZE, sizeof(int));
	int *d_result = 0;
	int i;
	
	cout << "a[N] array before scaling: [";
	for(i=0;i<ARRAY_SIZE;i++)
	{
		cout << oringinal[i] << " ";
	}
	cout << "]" << endl;
	
	hipMalloc((void**) &d_oringinal, sizeof(int)*ARRAY_SIZE);
	hipMemcpy(d_oringinal, oringinal, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &d_result, sizeof(int)*ARRAY_SIZE*8);
	hipMemcpy(d_result, result, sizeof(int)*ARRAY_SIZE*8, hipMemcpyHostToDevice);
	
    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    test<<<dimGrid,dimBlock>>>(d_oringinal, d_result);
	hipDeviceSynchronize();
	
	hipMemcpy(result, d_result,sizeof(int)*ARRAY_SIZE*8,hipMemcpyDeviceToHost);
	
	cout << "Result[N] array after scaling:"<<endl; 
    cout<<"[";
	for(i=0; i<ARRAY_SIZE; i++)
	{
		for(int j = 0; j < 8; j ++)
		{
			cout << result[8*i + j]<<",";
		}
        
	}
	cout << "]" << endl;
	
	return 0;
}
