#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
using namespace std;

const int BlockSize = 4;
const int ThreadNum = 4;
const int N = 4;
__global__ void MatAdd(int *A)
{
   int i = threadIdx.x;
   int j = threadIdx.y;
   printf("A[%d][%d]=%d\n", i, j, A[i * N + j]));
}


int main()
{
	int **a = (int **) malloc (N * sizeof(int *));
	for(int i=0; i < N;i++) 
		a[i]= (int *) malloc (N * sizeof(int));
	int *c_a;
	for (int i = 0; i < N; i++) 
		for(int j = 0; j < N; j++) 
			a[i][j] = 1;
	hipMalloc((void **)&c_a, N * sizeof(int) * N);
	hipMemcpy(c_a, a, N * sizeof(int) * N, hipMemcpyHostToDevice);

    dim3 dimBlock(BlockSize);
    dim3 dimGrid(ThreadNum);
    MatAdd<<<dimGrid,dimBlock>>>(c_a);

	return 0;
}
